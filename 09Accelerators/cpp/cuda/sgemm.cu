/*
 * CUDA SGEMM implementation.
 * Copied from https://github.com/garymacindoe/cuda-cholesky/blob/master/blas/sgemm.cu
 * 
 * Author: Gary Macindoe
 * Date: 02/05/2013
 * Modified for inclusion in Research Computing with C++ 31/03/2015
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

#define ITERATIONS 1000

#define CUDA_ERROR_CHECK(call) \
    do { \
        hipError_t error = (call); \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDART error in %s (%s:%d):\n\t%s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
            return error; \
        } \
    } while (false)

typedef enum { CBlasNoTrans = 'N', CBlasTrans = 'T', CBlasConjTrans = 'C' } CBlasTranspose;

/// "saxpy"
// y(1:16) += alpha * x(1:16)
__device__ void saxpy(float alpha, const float * __restrict__ x, float * __restrict__ y) {
  y[ 0] += alpha * x[ 0]; y[ 1] += alpha * x[ 1]; y[ 2] += alpha * x[ 2]; y[ 3] += alpha * x[ 3];
  y[ 4] += alpha * x[ 4]; y[ 5] += alpha * x[ 5]; y[ 6] += alpha * x[ 6]; y[ 7] += alpha * x[ 7];
  y[ 8] += alpha * x[ 8]; y[ 9] += alpha * x[ 9]; y[10] += alpha * x[10]; y[11] += alpha * x[11];
  y[12] += alpha * x[12]; y[13] += alpha * x[13]; y[14] += alpha * x[14]; y[15] += alpha * x[15];
}

/**
 * SGEMM:
 *   C := alpha * AB   + beta * C for transB == CBlasNoTrans
 *   C := alpha * AB'  + beta * C for transB == CBlasTrans
 *
 * @param transB  transpose for B.
 * @param mb      the number of rows in the block of C.
 * @param nb      the number of columns in the block of C.
 * @param kb      how far to unroll the inner loop.
 * @param bx      blockDim.x.
 * @param by      blockDim.y.
 */
template <CBlasTranspose transB,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void sgemm(const float * __restrict__ A, const float * __restrict__ B,
                      float * __restrict__ C,
                      float alpha, float beta,
                      int lda, int ldb, int ldc,
                      int m, int n, int k) {

  const int bi = blockIdx.x * mb;       // Starting row of block of C/C
  const int bj = blockIdx.y * nb;       // Starting column of block of C/C
  int ti = threadIdx.y * bx + threadIdx.x;
  int tj = 0;

  /*
   * Compute our starting points in A, B, C and C.
   *
   * For transA != CBlasNoTrans A is cached in shared memory so the unwrapped
   * thread index can be re-wrapped around mb when calculating C.
   *
   * If transA == CBlasNoTrans then bx * by == mb (checked later on) so there
   * doesn't need to be a separate check for transA == CBlasNoTrans in
   * calculating the start of C/C here.
   */
  A += bi + ti;
  B += (transB == CBlasNoTrans) ? (bj + threadIdx.y) * ldb + threadIdx.x : threadIdx.y * ldb + bj + threadIdx.x;
  C += (bj + tj) * ldc + bi + ti;
  n -= bj + tj;
  m -= bi + ti;

  /*
   * Block B in shared memory and C in registers.
   */
  __shared__ float b[kb][(transB == CBlasNoTrans) ? nb + 1 : nb];

  float c[] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

  while (k > 0) {
    // B will always be "transposed" w.r.t. C so must always be cached in shared
    // memory (i.e. it is read along the K or N dimensions when M is the
    // dimension being expanded).
    if (transB == CBlasNoTrans) {
#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * ldb];
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l += by) {
#pragma unroll
        for (int j = 0; j < nb; j += bx)
          b[l + threadIdx.y][j + threadIdx.x] = B[l * ldb + j];
      }
    }

    __syncthreads();

    if (k < kb) break;

    // Read A straight from global memory.
#pragma unroll
    for (int l = 0; l < kb; l++) {
      saxpy(A[0], b[l], c);
      A += lda;
    }

    __syncthreads();

    B += (transB == CBlasNoTrans) ? kb : kb * ldb;
    k -= kb;
  }

  for (int l = 0; l < k; l++) {
    saxpy(A[0], b[l], c);
    A += lda;
  }

  if (n <= 0 || m <= 0) return;
  if (beta == 0.0f) {
    C[0] = alpha * c[ 0]; if ( 1 >= n) return; C += ldc;
    C[0] = alpha * c[ 1]; if ( 2 >= n) return; C += ldc;
    C[0] = alpha * c[ 2]; if ( 3 >= n) return; C += ldc;
    C[0] = alpha * c[ 3]; if ( 4 >= n) return; C += ldc;
    C[0] = alpha * c[ 4]; if ( 5 >= n) return; C += ldc;
    C[0] = alpha * c[ 5]; if ( 6 >= n) return; C += ldc;
    C[0] = alpha * c[ 6]; if ( 7 >= n) return; C += ldc;
    C[0] = alpha * c[ 7]; if ( 8 >= n) return; C += ldc;
    C[0] = alpha * c[ 8]; if ( 9 >= n) return; C += ldc;
    C[0] = alpha * c[ 9]; if (10 >= n) return; C += ldc;
    C[0] = alpha * c[10]; if (11 >= n) return; C += ldc;
    C[0] = alpha * c[11]; if (12 >= n) return; C += ldc;
    C[0] = alpha * c[12]; if (13 >= n) return; C += ldc;
    C[0] = alpha * c[13]; if (14 >= n) return; C += ldc;
    C[0] = alpha * c[14]; if (15 >= n) return; C += ldc;
    C[0] = alpha * c[15];
  }
  else {
    C[0] = alpha * c[ 0] + beta * C[0]; if ( 1 >= n) return; C += ldc;
    C[0] = alpha * c[ 1] + beta * C[0]; if ( 2 >= n) return; C += ldc;
    C[0] = alpha * c[ 2] + beta * C[0]; if ( 3 >= n) return; C += ldc;
    C[0] = alpha * c[ 3] + beta * C[0]; if ( 4 >= n) return; C += ldc;
    C[0] = alpha * c[ 4] + beta * C[0]; if ( 5 >= n) return; C += ldc;
    C[0] = alpha * c[ 5] + beta * C[0]; if ( 6 >= n) return; C += ldc;
    C[0] = alpha * c[ 6] + beta * C[0]; if ( 7 >= n) return; C += ldc;
    C[0] = alpha * c[ 7] + beta * C[0]; if ( 8 >= n) return; C += ldc;
    C[0] = alpha * c[ 8] + beta * C[0]; if ( 9 >= n) return; C += ldc;
    C[0] = alpha * c[ 9] + beta * C[0]; if (10 >= n) return; C += ldc;
    C[0] = alpha * c[10] + beta * C[0]; if (11 >= n) return; C += ldc;
    C[0] = alpha * c[11] + beta * C[0]; if (12 >= n) return; C += ldc;
    C[0] = alpha * c[12] + beta * C[0]; if (13 >= n) return; C += ldc;
    C[0] = alpha * c[13] + beta * C[0]; if (14 >= n) return; C += ldc;
    C[0] = alpha * c[14] + beta * C[0]; if (15 >= n) return; C += ldc;
    C[0] = alpha * c[15] + beta * C[0];
  }
}

/**
 * For C = aAB + bC:
 *   mb must be a multiple of the warp size (32) and less than or equal to the
 *        maximum number of threads per block (512).
 *   nb must be less than or equal to 20 (registers start spilling to global
 *        memory after 20).
 *   kb must be a multiple of the half-warp size (16) and such that
 *        (nb + 1)*kb*sizeof(float) is less than the amount of shared memory
 *        available per block (16384 bytes).
 *
 * mb and nb must be selected such that the bandwidth reduction is greater than
 * the flop:word ratio of the GPU.  The bandwidth reduction for all valid values
 * of mb and nb can be calculated with the following loop (bash):
 * echo -n " mb\nb"; for nb in {1..20}; do printf "%6d" ${nb}; done; echo; for mb in {32..512..32}; do printf "%6d"  ${mb}; for nb in {1..20}; do printf "%6.2f" $(echo 2 / \(1/${mb} + 1/${nb}\) | bc -l); done; echo; done
 *
 * Sample output:
 *  mb\nb     1     2     3     4     5     6     7     8     9    10    11    12    13    14    15    16    17    18    19    20
 *     32  1.94  3.76  5.49  7.11  8.65 10.11 11.49 12.80 14.05 15.24 16.37 17.45 18.49 19.48 20.43 21.33 22.20 23.04 23.84 24.62
 *     64  1.97  3.88  5.73  7.53  9.28 10.97 12.62 14.22 15.78 17.30 18.77 20.21 21.61 22.97 24.30 25.60 26.86 28.10 29.30 30.48
 *     96  1.98  3.92  5.82  7.68  9.50 11.29 13.05 14.77 16.46 18.11 19.74 21.33 22.90 24.44 25.95 27.43 28.88 30.32 31.72 33.10
 *    128  1.98  3.94  5.86  7.76  9.62 11.46 13.27 15.06 16.82 18.55 20.26 21.94 23.60 25.24 26.85 28.44 30.01 31.56 33.09 34.59
 *    160  1.99  3.95  5.89  7.80  9.70 11.57 13.41 15.24 17.04 18.82 20.58 22.33 24.05 25.75 27.43 29.09 30.73 32.36 33.97 35.56
 *    192  1.99  3.96  5.91  7.84  9.75 11.64 13.51 15.36 17.19 19.01 20.81 22.59 24.35 26.10 27.83 29.54 31.23 32.91 34.58 36.23
 *    224  1.99  3.96  5.92  7.86  9.78 11.69 13.58 15.45 17.30 19.15 20.97 22.78 24.57 26.35 28.12 29.87 31.60 33.32 35.03 36.72
 *    256  1.99  3.97  5.93  7.88  9.81 11.73 13.63 15.52 17.39 19.25 21.09 22.93 24.74 26.55 28.34 30.12 31.88 33.64 35.37 37.10
 *    288  1.99  3.97  5.94  7.89  9.83 11.76 13.67 15.57 17.45 19.33 21.19 23.04 24.88 26.70 28.51 30.32 32.10 33.88 35.65 37.40
 *    320  1.99  3.98  5.94  7.90  9.85 11.78 13.70 15.61 17.51 19.39 21.27 23.13 24.98 26.83 28.66 30.48 32.28 34.08 35.87 37.65
 *    352  1.99  3.98  5.95  7.91  9.86 11.80 13.73 15.64 17.55 19.45 21.33 23.21 25.07 26.93 28.77 30.61 32.43 34.25 36.05 37.85
 *    384  1.99  3.98  5.95  7.92  9.87 11.82 13.75 15.67 17.59 19.49 21.39 23.27 25.15 27.02 28.87 30.72 32.56 34.39 36.21 38.02
 *    416  2.00  3.98  5.96  7.92  9.88 11.83 13.77 15.70 17.62 19.53 21.43 23.33 25.21 27.09 28.96 30.81 32.67 34.51 36.34 38.17
 *    448  2.00  3.98  5.96  7.93  9.89 11.84 13.78 15.72 17.65 19.56 21.47 23.37 25.27 27.15 29.03 30.90 32.76 34.61 36.45 38.29
 *    480  2.00  3.98  5.96  7.93  9.90 11.85 13.80 15.74 17.67 19.59 21.51 23.41 25.31 27.21 29.09 30.97 32.84 34.70 36.55 38.40
 *    512  2.00  3.98  5.97  7.94  9.90 11.86 13.81 15.75 17.69 19.62 21.54 23.45 25.36 27.25 29.15 31.03 32.91 34.78 36.64 38.50
 *
 * The number of registers per block is mb*32 (compiled with -maxrregcount=32).
 * More threads == better performance (from flop-test) therefore mb is chosen to
 * be the largest number of threads such that the number of blocks per
 * multiprocessor is still limited by the register usage.
 * kb is chosen to be the largest multiple of 16 such that the number of blocks
 * per multiprocessor is limited by the register usage.
 */

/// "main"
int main(int argc, char * argv[]) {
    int m = 320, n = 640, k = 640;

    if (argc != 1 && argc != 4) {
        fprintf(stderr, "Usage: %s [m=320 n=640 k=640]\n", argv[0]);
        return -1;
    }

    if (argc == 4) {
		if (sscanf(argv[1], "%d", &m) != 1) {
			fprintf(stderr, "Failed to parse m from '%s'\n", argv[1]);
			return -2;
		}

		if (sscanf(argv[2], "%d", &n) != 1) {
			fprintf(stderr, "Failed to parse n from '%s'\n", argv[2]);
			return -3;
		}

		if (sscanf(argv[3], "%d", &k) != 1) {
			fprintf(stderr, "Failed to parse k from '%s'\n", argv[3]);
			return -4;
		}
    }

    printf("m = %d, n = %d, k = %d\n", m, n, k);

    float a = 0.5, * A, * B, b = 1.2, * C;

    // Round matrix column lengths up to multiple of SIMD width so each column
    // is correctly aligned in memory.  The value 3 is calculated as SIMD width /
    // sizeof(type) - 1 which for single precision floats using SSE is 128 / 32
    // - 1.  This does for host memory what cudaMallocPitch does for device memory.
    size_t lda = ((unsigned int)m + 3u) & ~3u;
    size_t ldb = ((unsigned int)k + 3u) & ~3u;
    size_t ldc = ((unsigned int)m + 3u) & ~3u;

    CUDA_ERROR_CHECK(hipHostMalloc(&A, lda * k * sizeof(float), hipHostMallocDefault));
    CUDA_ERROR_CHECK(hipHostMalloc(&B, ldb * n * sizeof(float), hipHostMallocDefault));
    CUDA_ERROR_CHECK(hipHostMalloc(&C, ldc * n * sizeof(float), hipHostMallocDefault));

    // Initialise A, B and C ~U(0,1)
	for (size_t j = 0; j < k; j++) {
		for (size_t i = 0; i < m; i++)
    		A[j * lda + i] = (float)rand() / RAND_MAX;
    }
	for (size_t j = 0; j < n; j++) {
		for (size_t i = 0; i < k; i++)
    		B[j * ldb + i] = (float)rand() / RAND_MAX;
    }
	for (size_t j = 0; j < n; j++) {
		for (size_t i = 0; i < m; i++)
    		C[j * ldc + i] = (float)rand() / RAND_MAX;
    }

	/// "cublas_sgemm"
	// Allocate matrices on GPU
    float * dA, * dB, * dC;
    size_t dlda, dldb, dldc;
    CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, m * sizeof(float), k));
    CUDA_ERROR_CHECK(hipMallocPitch((void **)&dB, &dldb, k * sizeof(float), n));
    CUDA_ERROR_CHECK(hipMallocPitch((void **)&dC, &dldc, m * sizeof(float), n));

    // cudaMallocPitch returns leading dimensions in bytes while SGEMM expects
    // them as number of elements
    dlda /= sizeof(float);
    dldb /= sizeof(float);
    dldc /= sizeof(float);

    CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(float), A, lda * sizeof(float), m * sizeof(float), k, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy2D(dB, dldb * sizeof(float), B, ldb * sizeof(float), k * sizeof(float), n, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy2D(dC, dldc * sizeof(float), C, ldc * sizeof(float), m * sizeof(float), n, hipMemcpyHostToDevice));

    float gpu_total = 0.0f;
    hipEvent_t start, end;
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&end));
    for (int i = 0; i < ITERATIONS; i++) {
        CUDA_ERROR_CHECK(hipEventRecord(start, NULL));

        dim3 blocks((m + 63) / 64, (n + 15) / 16);
        dim3 threads(16, 16);
        sgemm<CBlasNoTrans, 64, 16, 16, 16,  4><<<blocks, threads>>>(dA, dB, dC, a, b, dlda, dldb, dldc, m, n, k);

        CUDA_ERROR_CHECK(hipEventRecord(end, NULL));
        CUDA_ERROR_CHECK(hipEventSynchronize(end));
        float t;
        CUDA_ERROR_CHECK(hipEventElapsedTime(&t, start, end));
        gpu_total += t;
    }
    CUDA_ERROR_CHECK(hipEventDestroy(start));
    CUDA_ERROR_CHECK(hipEventDestroy(end));

    // Free memory
    CUDA_ERROR_CHECK(hipFree(dA));
    CUDA_ERROR_CHECK(hipFree(dB));
    CUDA_ERROR_CHECK(hipFree(dC));
    CUDA_ERROR_CHECK(hipHostFree(A));
    CUDA_ERROR_CHECK(hipHostFree(B));
    CUDA_ERROR_CHECK(hipHostFree(C));

    // Print results
    double gpu_time = gpu_total / (1000 * ITERATIONS);
    double bandwidth = 2 * m * n * k * sizeof(float);
    double flops = 2 * m * n * k;
    printf("Bandwidth: %.3fGB/s\n", (bandwidth / gpu_time) / 1.e9);
    printf("Throughput: %.3fGFlops/s\n", (flops / gpu_time) / 1.e9);

    return 0;
}
