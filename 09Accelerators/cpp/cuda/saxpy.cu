#include <stdio.h>
#include <hip/hip_runtime.h>

#define ITERATIONS 1000

#define CUDA_ERROR_CHECK(call) \
    do { \
        hipError_t error = (call); \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDART error in %s (%s:%d):\n\t%s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
            return error; \
        } \
    } while (false)

/// "saxpy"
__global__ void saxpy(int n, float a, const float * __restrict__ x, int incx,
                      float * __restrict__ y, int incy) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] += a * x[i];
}

/// "main"
int main(int argc, char * argv[]) {
    int n = 10000, incx = 1, incy = 1;

    if (argc < 1 || argc > 4) {
        fprintf(stderr, "Usage: %s [n=10000 [incx=1 [incy=1]]]\n", argv[0]);
        return -1;
    }

    if (argc > 1) {
        if (sscanf(argv[1], "%d", &n) != 1) {
            fprintf(stderr, "Failed to parse n from '%s'\n", argv[1]);
            return -2;
        }

        if (argc > 2) {
            if (sscanf(argv[2], "%d", &incx) != 1) {
                fprintf(stderr, "Failed to parse incx from '%s'\n", argv[2]);
                return -2;
            }

            if (argc > 3) {
                if (sscanf(argv[3], "%d", &incy) != 1) {
                    fprintf(stderr, "Failed to parse incy from '%s'\n", argv[3]);
                    return -3;
                }
            }
        }
    }

    printf("n = %d, incx = %d, incy = %d\n", n, incx, incy);

    const float a = 1.5f;
    float * x, * y, * dx, * dy;
    CUDA_ERROR_CHECK(hipHostMalloc(&x, n * incx * sizeof(float), hipHostMallocDefault));
    CUDA_ERROR_CHECK(hipHostMalloc(&y, n * incy * sizeof(float), hipHostMallocDefault));
    CUDA_ERROR_CHECK(hipMalloc(&dx, n * incx * sizeof(float)));
    CUDA_ERROR_CHECK(hipMalloc(&dy, n * incy * sizeof(float)));

    for (int i = 0; i < n; i++) {
        x[i * incx] = rand() / RAND_MAX;
        y[i * incy] = rand() / RAND_MAX;
    }
    
    CUDA_ERROR_CHECK(hipMemcpy(dx, x, n * incx * sizeof(float), hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy(dy, y, n * incy * sizeof(float), hipMemcpyHostToDevice));

    float total = 0.0f;
    hipEvent_t start, end;
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&end));
    for (int i = 0; i < ITERATIONS; i++) {
        CUDA_ERROR_CHECK(hipEventRecord(start, NULL));
/// "CudaCall"
        saxpy<<<n/64 + 1, 64>>>(n, a, dx, incx, dy, incy);
/// "CudaCalled"
        CUDA_ERROR_CHECK(hipEventRecord(end, NULL));
        CUDA_ERROR_CHECK(hipEventSynchronize(end));
        float t;
        CUDA_ERROR_CHECK(hipEventElapsedTime(&t, start, end));
        total += t;
    }
    CUDA_ERROR_CHECK(hipEventDestroy(start));
    CUDA_ERROR_CHECK(hipEventDestroy(end));
    printf("saxpy: %.6fms\n", total/ITERATIONS);

    CUDA_ERROR_CHECK(hipHostFree(x));
    CUDA_ERROR_CHECK(hipHostFree(y));
    CUDA_ERROR_CHECK(hipFree(dx));
    CUDA_ERROR_CHECK(hipFree(dy));

    return 0;
}
